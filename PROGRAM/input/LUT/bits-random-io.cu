#include "hip/hip_runtime.h"
#include "bits-random.h"

//IO

void lee_betas(char * name_betas)
{
  int ibeta;

  double dummy;
  FILE *fi_betas;
  

  
  if (NULL==(fi_betas=fopen(name_betas,"rt")))
    print_and_exit(" No existe el fichero %s.\n",name_betas);

  for (ibeta=0;ibeta<NUMBETAS;ibeta++)
    if (EOF==fscanf(fi_betas,"%lf ",&betas[ibeta]))
      print_and_exit("El fichero %s solo tiene %d lineas (necesita %d)\n",
		     name_betas,ibeta,NUMBETAS);
  // hay alguna beta de sobra en el fichero?
  if (EOF!=fscanf(fi_betas,"%lf",&dummy))
    fprintf(stderr,"Hay lineas de mas en el fichero %s\n",name_betas); 
  
  for(ibeta=0;ibeta<NUMBETAS-1;ibeta++){
    if (betas[ibeta]<betas[ibeta+1])
      print_and_exit("Error en fichero %s: betas no ordenadas de mayor a menor!\n",name_betas);
  }

  fclose(fi_betas);
}


void lee_lut(char * name_betas)  
{
  FILE *Fin;
  char name[1024];
  double betas_read[NUMBETAS];
  int ib;
  
  sprintf(name,"LUT_for_PRNG_nbits%02d_NB%d.bin",NUMBITSPREBUSQUEDAS,NUMBETAS);
  if(NULL==(Fin=fopen(name,"rb")))
    print_and_exit("Problemas abriendo %s\n",name);

  for(ib=0;ib<NUMBETAS;ib++)
    betas_read[ib]=0;
  
  if(1!=fread(betas_read,sizeof(betas_read),(size_t) 1,Fin)){
    printf("betas_read[0]=%.14g\n",betas_read[0]);
    printf("betas_read[1]=%.14g\n",betas_read[1]);
    print_and_exit("problemas leyendo betas\n");
  }
  if(1!=fread(lut_heat_bath,sizeof(lut_heat_bath),(size_t) 1,Fin))
    print_and_exit("problemas leyendo lut_heat_bath\n");

  for(ib=0;ib<NUMBETAS;ib++){
    if(fabs(betas[ib]-betas_read[ib])>1e-14)
      print_and_exit("betas[%d]=%.14g from %s, betas[%d]=%.14g from %s (difference=%.14g)\n",
		     ib,betas[ib],name_betas,ib,betas_read[ib],name,betas[ib]-betas_read[ib]);
  }
  
  fclose(Fin);
}


void print_and_exit(const char *format, ...)
{
  va_list list;
    
  va_start(list,format);
  vfprintf(stderr,format,list);
  va_end(list);
  exit(1);
}


//Rutina tomada de "GNU radio"
void obten_crc(uint32_t *crc_thread_sums)
{

  static const uint32_t table_crc[256] = {
    0x00000000U,0x04C11DB7U,0x09823B6EU,0x0D4326D9U,
    0x130476DCU,0x17C56B6BU,0x1A864DB2U,0x1E475005U,
    0x2608EDB8U,0x22C9F00FU,0x2F8AD6D6U,0x2B4BCB61U,
    0x350C9B64U,0x31CD86D3U,0x3C8EA00AU,0x384FBDBDU,
    0x4C11DB70U,0x48D0C6C7U,0x4593E01EU,0x4152FDA9U,
    0x5F15ADACU,0x5BD4B01BU,0x569796C2U,0x52568B75U,
    0x6A1936C8U,0x6ED82B7FU,0x639B0DA6U,0x675A1011U,
    0x791D4014U,0x7DDC5DA3U,0x709F7B7AU,0x745E66CDU,
    0x9823B6E0U,0x9CE2AB57U,0x91A18D8EU,0x95609039U,
    0x8B27C03CU,0x8FE6DD8BU,0x82A5FB52U,0x8664E6E5U,
    0xBE2B5B58U,0xBAEA46EFU,0xB7A96036U,0xB3687D81U,
    0xAD2F2D84U,0xA9EE3033U,0xA4AD16EAU,0xA06C0B5DU,
    0xD4326D90U,0xD0F37027U,0xDDB056FEU,0xD9714B49U,
    0xC7361B4CU,0xC3F706FBU,0xCEB42022U,0xCA753D95U,
    0xF23A8028U,0xF6FB9D9FU,0xFBB8BB46U,0xFF79A6F1U,
    0xE13EF6F4U,0xE5FFEB43U,0xE8BCCD9AU,0xEC7DD02DU,
    0x34867077U,0x30476DC0U,0x3D044B19U,0x39C556AEU,
    0x278206ABU,0x23431B1CU,0x2E003DC5U,0x2AC12072U,
    0x128E9DCFU,0x164F8078U,0x1B0CA6A1U,0x1FCDBB16U,
    0x018AEB13U,0x054BF6A4U,0x0808D07DU,0x0CC9CDCAU,
    0x7897AB07U,0x7C56B6B0U,0x71159069U,0x75D48DDEU,
    0x6B93DDDBU,0x6F52C06CU,0x6211E6B5U,0x66D0FB02U,
    0x5E9F46BFU,0x5A5E5B08U,0x571D7DD1U,0x53DC6066U,
    0x4D9B3063U,0x495A2DD4U,0x44190B0DU,0x40D816BAU,
    0xACA5C697U,0xA864DB20U,0xA527FDF9U,0xA1E6E04EU,
    0xBFA1B04BU,0xBB60ADFCU,0xB6238B25U,0xB2E29692U,
    0x8AAD2B2FU,0x8E6C3698U,0x832F1041U,0x87EE0DF6U,
    0x99A95DF3U,0x9D684044U,0x902B669DU,0x94EA7B2AU,
    0xE0B41DE7U,0xE4750050U,0xE9362689U,0xEDF73B3EU,
    0xF3B06B3BU,0xF771768CU,0xFA325055U,0xFEF34DE2U,
    0xC6BCF05FU,0xC27DEDE8U,0xCF3ECB31U,0xCBFFD686U,
    0xD5B88683U,0xD1799B34U,0xDC3ABDEDU,0xD8FBA05AU,
    0x690CE0EEU,0x6DCDFD59U,0x608EDB80U,0x644FC637U,
    0x7A089632U,0x7EC98B85U,0x738AAD5CU,0x774BB0EBU,
    0x4F040D56U,0x4BC510E1U,0x46863638U,0x42472B8FU,
    0x5C007B8AU,0x58C1663DU,0x558240E4U,0x51435D53U,
    0x251D3B9EU,0x21DC2629U,0x2C9F00F0U,0x285E1D47U,
    0x36194D42U,0x32D850F5U,0x3F9B762CU,0x3B5A6B9BU,
    0x0315D626U,0x07D4CB91U,0x0A97ED48U,0x0E56F0FFU,
    0x1011A0FAU,0x14D0BD4DU,0x19939B94U,0x1D528623U,
    0xF12F560EU,0xF5EE4BB9U,0xF8AD6D60U,0xFC6C70D7U,
    0xE22B20D2U,0xE6EA3D65U,0xEBA91BBCU,0xEF68060BU,
    0xD727BBB6U,0xD3E6A601U,0xDEA580D8U,0xDA649D6FU,
    0xC423CD6AU,0xC0E2D0DDU,0xCDA1F604U,0xC960EBB3U,
    0xBD3E8D7EU,0xB9FF90C9U,0xB4BCB610U,0xB07DABA7U,
    0xAE3AFBA2U,0xAAFBE615U,0xA7B8C0CCU,0xA379DD7BU,
    0x9B3660C6U,0x9FF77D71U,0x92B45BA8U,0x9675461FU,
    0x8832161AU,0x8CF30BADU,0x81B02D74U,0x857130C3U,
    0x5D8A9099U,0x594B8D2EU,0x5408ABF7U,0x50C9B640U,
    0x4E8EE645U,0x4A4FFBF2U,0x470CDD2BU,0x43CDC09CU,
    0x7B827D21U,0x7F436096U,0x7200464FU,0x76C15BF8U,
    0x68860BFDU,0x6C47164AU,0x61043093U,0x65C52D24U,
    0x119B4BE9U,0x155A565EU,0x18197087U,0x1CD86D30U,
    0x029F3D35U,0x065E2082U,0x0B1D065BU,0x0FDC1BECU,
    0x3793A651U,0x3352BBE6U,0x3E119D3FU,0x3AD08088U,
    0x2497D08DU,0x2056CD3AU,0x2D15EBE3U,0x29D4F654U,
    0xC5A92679U,0xC1683BCEU,0xCC2B1D17U,0xC8EA00A0U,
    0xD6AD50A5U,0xD26C4D12U,0xDF2F6BCBU,0xDBEE767CU,
    0xE3A1CBC1U,0xE760D676U,0xEA23F0AFU,0xEEE2ED18U,
    0xF0A5BD1DU,0xF464A0AAU,0xF9278673U,0xFDE69BC4U,
    0x89B8FD09U,0x8D79E0BEU,0x803AC667U,0x84FBDBD0U,
    0x9ABC8BD5U,0x9E7D9662U,0x933EB0BBU,0x97FFAD0CU,
    0xAFB010B1U,0xAB710D06U,0xA6322BDFU,0xA2F33668U,
    0xBCB4666DU,0xB8757BDAU,0xB5365D03U,0xB1F740B4U};
   
  uint8_t data;
  uint32_t temporal,crc,total;
  int indice,ibyte,barrera;
  union{
    uint4 palabra;
    uint32_t vec[4];
  }sumas;

  total=MSC_V*NUMBETAS;
    
  for(barrera=0;barrera<3;barrera++)
    crc_thread_sums[barrera]=0xffffffffU;

  for(indice=0;indice<total;indice++){
    sumas.palabra=suma_global[indice];
    for(barrera=0;barrera<3;barrera++){
      temporal=sumas.vec[barrera];
      crc=crc_thread_sums[barrera];
      for(ibyte=0;ibyte<4;ibyte++){
	data=(uint8_t) (temporal&255U);
	temporal>>=8;
	crc = table_crc[data ^ ((crc >> 24) & 0xff)] ^ (crc << 8);
      }
      crc_thread_sums[barrera]=crc; 
    }
  }
      
}

void comprueba_resultados(long long int num_pasos,uint64_t semilla, uint32_t semilla_key)
{
  int indice,ib,site,barrera;
  double beta;
  
  double prob[3],error,empirical_prob,total;
  double esperado[3],dispersion[3];
  unsigned long long suma[3];
  uint32_t crc_thread_sums[3];
  union{
    uint4 palabra;
    uint32_t vec[4];
  }cuenta;
  FILE * Fout;

  if(NULL==(Fout=fopen("comprobacion_resultados_GPU.txt","wt")))
    print_and_exit("Problemas abriendo comprobacion_resultados_GPU.txt");
  
  fprintf(Fout,"Comprobando el generador en la GPU con semilla %llu\n", (unsigned long long) semilla);
  fprintf(Fout,"semilla_key=%u (binario:%u%u%u%u...)\n",semilla_key,(semilla_key>>31)&1U,
	 (semilla_key>>30)&1U,(semilla_key>>29)&1U,(semilla_key>>28)&1U);

  

  obten_crc(crc_thread_sums);

  indice=0;
  for(ib=0;ib<NUMBETAS;ib++){
    beta=betas[ib];
    for(barrera=0;barrera<3;barrera++){
      prob[barrera]=exp(-4.*beta*(barrera+1));
      esperado[barrera]=prob[barrera]*64.*((double) num_pasos);
      dispersion[barrera]=sqrt(prob[barrera]*(1.-prob[barrera])*64.*((double) num_pasos));
    }
    suma[0]=0; suma[1]=0;suma[2]=0;
    for(site=0;site<MSC_V;site++){
      cuenta.palabra=suma_global[indice];
      suma[0]+=suma_global[indice].x;
      suma[1]+=suma_global[indice].y;
      suma[2]+=suma_global[indice].z;
      for(barrera=0;barrera<3;barrera++){
	if(fabs((cuenta.vec[barrera]-esperado[barrera])/dispersion[barrera])>4.){
	  fprintf(Fout,"Gran desviacion ib=%d  site=%d barrera=%d: obtengo %u y espero %.14g (%.14g sigma)\n",
		 ib,site,4*(barrera+1),cuenta.vec[barrera],esperado[barrera],
		 (cuenta.vec[barrera]-esperado[barrera])/dispersion[barrera]);
	}
      }
      indice++;
    }
    total=((double) num_pasos*64.*MSC_V);

    for(barrera=0;barrera<3;barrera++){ 
      error=sqrt(prob[barrera]*(1-prob[barrera])/total);
      empirical_prob=((double) suma[barrera])/total;
      fprintf(Fout,"Densidad de bits en b_%d=%.14g (exact: %.14g, error=%.14g sigma) ib=%d\n",
	     4*(barrera+1),
	     empirical_prob,prob[barrera],(empirical_prob-prob[barrera])/error,
	     ib);
    }
  }
  fprintf(Fout,"Checksums:\n");
  for(barrera=0;barrera<3;barrera++){
    fprintf(Fout,"Barrera %d: suma_total=%llu,\t CRC-32 de las suma_global[tid]=%u\n",4*(barrera+1),
	   suma[barrera],crc_thread_sums[barrera]);
  }

  fclose(Fout);
}
